#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>



// Обычная реализация
__global__ void matmulsimp(int* M, int* N, int* P, int width) {
	for (int row = 0; row < width; row++) {
		for (int col = 0; col < width; col++) {
			// Multiply the row of A by the column of B to get the row, column of product.
			for (int inner = 0; inner < width; inner++)
				P[row * width + col] += M[row * width + inner] * N[inner * width + col];
		}
	}
}


// Параллельная
__global__ void MatMul(int* M, int* N, int* P, int width)
{
	int x = threadIdx.x;
	int y = threadIdx.y;

	float Pervalue = 0;

	float elem1 = 0.0, elem2 = 0.0, value = 0.0;
	for (int i = 0; i < width; i++)
	{
		elem1 = M[y * width + i]; // Возьмем одну строку матрицы M
		elem2 = N[i * width + x]; // Берем один столбец из N матрицы

		value += elem1 * elem2; // Сумма
	}

	P[y * width + x] = value;
}

int main()
{
	const int ND = 100;
	int a[ND][ND], b[ND][ND], c[ND][ND];
	int *M, *N, *P;

	int width = ND;
	dim3 blockSize(ND, ND);

	hipEvent_t start, stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Распределение памяти устройства
	hipMalloc((void**)&M, ND * ND * sizeof(int));
	hipMalloc((void**)&N, ND * ND * sizeof(int));
	hipMalloc((void**)&P, ND * ND * sizeof(int));

	// Инициализация
	for (int i = 0; i < ND; i++)
	{
		for (int j = 0; j < ND; j++)
		{
			a[i][j] = 2;
			b[i][j] = 3;
		}
	}

	int Size = ND * ND;
	// Копирование данных с хоста на устройство
	hipMemcpy(M, a, Size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(N, b, Size * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	MatMul <<< 1, blockSize >>> (M, N, P, width); // Вызов функции ядра
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(c, P, Size * sizeof(int), hipMemcpyDeviceToHost);

	printf("c0 = %f \n", elapsedTime);

	hipEventRecord(start, 0);
	matmulsimp <<<1, 1>>>(M, N, P, ND);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);


	printf("c0 = %f \n", elapsedTime);

	// Освободить память устройства
	hipFree(M);
	hipFree(N);
	hipFree(P);

	return 0;
}